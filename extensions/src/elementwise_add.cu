#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <stdio.h>

#define BLOCK_SIZE 32

__global__ void add(unsigned const int M, unsigned const int N, const float* A, const float* B, float* C) {
    int row = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int col = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int array_idx = (row * N) + col;
    if (row < M && col < N) {
        C[ array_idx ] = A[array_idx] + B[array_idx];
    }
}

void elementwise_add(unsigned const int M, unsigned const int N, const float* A, const float* B, float* C) {
    dim3 dimBlocks(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid( (N / BLOCK_SIZE) + 1, (M / BLOCK_SIZE) + 1);
    add<<< dimGrid, dimBlocks >>>(M, N, A, B, C);
    hipDeviceSynchronize();
}