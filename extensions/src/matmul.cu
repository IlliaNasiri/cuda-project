#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <stdio.h>

#define BLOCK_SIZE 32


/**
 * CUDA kernel that multiplies matrix A (MxK) by matrix B (KxN), and stores result in matrix C (MxN)
 * @param M number of rows in left matrix A, and number of rows resulting matrix result C
 * @param N number of columns in right matrix A, and number of columns in matrix result C
 * @param K number of columns in matrix A, number of rows in matrix B
 * @param A float32 array located on the GPU memory, that represents flattened matrix A
 * @param B float32 array located on the GPU memory, that represents flattened matrix B
 * @param C float32 array located on the GPU memory, that represents flattened result matrix C
 */
__global__ void sgemm(unsigned const int M, unsigned const int N, unsigned const int K, const float* A, const float* B, float* C) {
    __shared__ float A_shared[BLOCK_SIZE * BLOCK_SIZE];
    __shared__ float B_shared[BLOCK_SIZE * BLOCK_SIZE];

    unsigned int
        block_row = threadIdx.x / BLOCK_SIZE,
        block_col = threadIdx.x % BLOCK_SIZE,
        global_row = blockIdx.y * BLOCK_SIZE + block_row,
        global_col = blockIdx.x * BLOCK_SIZE + block_col,
        stride = 0;

    float tmp = 0.0f;

    for(int i = 0; i < K; i += BLOCK_SIZE) {
        unsigned int A_shared_idx = block_row * BLOCK_SIZE + block_col;
        unsigned int B_shared_idx = block_row * BLOCK_SIZE + block_col;

        A_shared[A_shared_idx] = ((stride + block_col) < K) ?
                A[(global_row * K) + stride + block_col] : 0.0f;

        B_shared[B_shared_idx] = ((stride + block_row) < K) ?
                B[(stride + block_row) * N + global_col] : 0.0f;

        __syncthreads();

        for(int j = 0; j < BLOCK_SIZE; j++)
            tmp += A_shared[block_row * BLOCK_SIZE + j] * B_shared[j * BLOCK_SIZE + block_col];

        __syncthreads();
        stride += BLOCK_SIZE;
    }

    if(global_row < M && global_col < N )
        C[global_row * N + global_col] = tmp;



}

void matmul(unsigned const int M, unsigned const int N, unsigned const int K, const float* A, const float* B, float* C) {
    dim3 dimBlocks(BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimGrid( (N / BLOCK_SIZE) + 1, (M / BLOCK_SIZE) + 1);
    sgemm<<< dimGrid, dimBlocks >>>(M, N, K, A, B, C);
    hipDeviceSynchronize();
}



